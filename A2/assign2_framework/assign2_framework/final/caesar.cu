#include "hip/hip_runtime.h"
/*
 * Names: Thijs van Solt, Fedja Matti
 * Student IDS: 13967681, 13953699
 * BSc Computer Science UvA
 * Description: This file contains an GPU version for
 *              encryption and decryption using
                Caesar and Vigenere.
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <iostream>
#include<ctype.h>

#include "file.hh"
#include "timer.hh"

using namespace std;

__constant__ int file_size;
__constant__ int length_key;

/* Utility function, use to do error checking for CUDA calls
 *
 * Use this function like this:
 *     checkCudaCall(<cuda_call>);
 *
 * For example:
 *     checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));
 *
 * Special case to check the result of the last kernel invocation:
 *     kernel<<<...>>>(...);
 *     checkCudaCall(hipGetLastError());
**/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(EXIT_FAILURE);
    }
}

// Encrypts a single character using Caesar
__global__ void encryptKernel(char* deviceDataIn, char* deviceDataOut, int* key) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < file_size) {
        deviceDataOut[i] = (deviceDataIn[i] + key[i % length_key]) % 256;
    }
}

// Decrypts a single character using Caesar
__global__ void decryptKernel(char* deviceDataIn, char* deviceDataOut, int* key) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < file_size) {
        deviceDataOut[i] = (deviceDataIn[i] - key[i % length_key]) % 256;
    }
}

// Encrypts a file using Caesar or Vigenere
int EncryptSeq (int n, char* data_in, char* data_out, int key_length, int *key) {
    timer sequentialTime = timer("Sequential encryption");

    sequentialTime.start();
    for (int i = 0; i < n; i++) {
        data_out[i] = (data_in[i] + key[i % key_length]) % 256;
    }
    sequentialTime.stop();

    cout << fixed << setprecision(6);
    cout << "Encryption (sequential): \t\t" << sequentialTime.getElapsed() << " seconds." << endl;

    return 0;
}

// Decrypts a file using Caesar or Vigenere
int DecryptSeq (int n, char* data_in, char* data_out, int key_length, int *key)
{
    timer sequentialTime = timer("Sequential decryption");

    sequentialTime.start();
    for (int i = 0; i < n; i++) {
        data_out[i] = (data_in[i] - key[i % key_length]) % 256;
    }
    sequentialTime.stop();

    cout << fixed << setprecision(6);
    cout << "Decryption (sequential): \t\t" << sequentialTime.getElapsed() << " seconds." << endl;

    return 0;
}

/* Wrapper for your encrypt kernel, i.e., does the necessary preparations and
 * calls your kernel. */
int EncryptCuda (int n, char* data_in, char* data_out, int key_length, int *key) {
    int threadBlockSize = 512;
    checkCudaCall(hipMemcpyToSymbol(HIP_SYMBOL(length_key), &key_length, sizeof(int)));
    checkCudaCall(hipMemcpyToSymbol(HIP_SYMBOL(file_size), &n, sizeof(int)));

    int* deviceKey = NULL;
    checkCudaCall(hipMalloc((void **) &deviceKey, key_length * sizeof(int)));
    if (deviceKey == NULL) {
        cerr << "Error allocating device memory for key" << endl;
        exit(EXIT_FAILURE);
    }
    // allocate the vectors on the GPU
    char* deviceDataIn = NULL;
    checkCudaCall(hipMalloc((void **) &deviceDataIn, n * sizeof(char)));
    if (deviceDataIn == NULL) {
        checkCudaCall(hipFree(deviceKey));
        cout << "could not allocate memory!" << endl;
        return -1;
    }
    char* deviceDataOut = NULL;
    checkCudaCall(hipMalloc((void **) &deviceDataOut, n * sizeof(char)));
    if (deviceDataOut == NULL) {
        checkCudaCall(hipFree(deviceDataIn));
        cout << "could not allocate memory!" << endl;
        return -1;
    }

    timer kernelTime1 = timer("kernelTime");
    timer memoryTime = timer("memoryTime");

    // copy the original vectors to the GPU
    memoryTime.start();
    checkCudaCall(hipMemcpy(deviceDataIn, data_in, n*sizeof(char), hipMemcpyHostToDevice));
    checkCudaCall(hipMemcpy(deviceKey, key, key_length*sizeof(int), hipMemcpyHostToDevice));
    memoryTime.stop();

    // execute kernel
    kernelTime1.start();
    if (n % threadBlockSize == 0) {
        encryptKernel<<<n/threadBlockSize, threadBlockSize>>>(deviceDataIn, deviceDataOut, deviceKey);
    } else {
        encryptKernel<<<(n/threadBlockSize) + 1, threadBlockSize>>>(deviceDataIn, deviceDataOut, deviceKey);
    }
    hipDeviceSynchronize();
    kernelTime1.stop();

    // check whether the kernel invocation was successful
    checkCudaCall(hipGetLastError());

    // copy result back
    memoryTime.start();
    checkCudaCall(hipMemcpy(data_out, deviceDataOut, n * sizeof(char), hipMemcpyDeviceToHost));
    memoryTime.stop();

    checkCudaCall(hipFree(deviceDataIn));
    checkCudaCall(hipFree(deviceDataOut));
    checkCudaCall(hipFree(deviceKey));

    cout << fixed << setprecision(6);
    cout << "Encrypt (kernel): \t\t" << kernelTime1.getElapsed() << " seconds." << endl;
    cout << "Encrypt (memory): \t\t" << memoryTime.getElapsed() << " seconds." << endl;

   return 0;
}

/* Wrapper for your decrypt kernel, i.e., does the necessary preparations and
 * calls your kernel. */
int DecryptCuda (int n, char* data_in, char* data_out, int key_length, int *key) {
    int threadBlockSize = 512;
    checkCudaCall(hipMemcpyToSymbol(HIP_SYMBOL(length_key), &key_length, sizeof(int)));
    checkCudaCall(hipMemcpyToSymbol(HIP_SYMBOL(file_size), &n, sizeof(int)));

    int* deviceKey = NULL;
    checkCudaCall(hipMalloc((void **) &deviceKey, key_length * sizeof(int)));
    if (deviceKey == NULL) {
        cerr << "Error allocating device memory for key" << endl;
        exit(EXIT_FAILURE);
    }

    // allocate the vectors on the GPU
    char* deviceDataIn = NULL;
    checkCudaCall(hipMalloc((void **) &deviceDataIn, n * sizeof(char)));
    if (deviceDataIn == NULL) {
        checkCudaCall(hipFree(deviceKey));
        cout << "could not allocate memory!" << endl;
        return -1;
    }
    char* deviceDataOut = NULL;
    checkCudaCall(hipMalloc((void **) &deviceDataOut, n * sizeof(char)));
    if (deviceDataOut == NULL) {
        checkCudaCall(hipFree(deviceDataIn));
        cout << "could not allocate memory!" << endl;
        return -1;
    }

    timer kernelTime1 = timer("kernelTime");
    timer memoryTime = timer("memoryTime");

    // copy the original vectors to the GPU
    memoryTime.start();
    checkCudaCall(hipMemcpy(deviceDataIn, data_in, n*sizeof(char), hipMemcpyHostToDevice));
    checkCudaCall(hipMemcpy(deviceKey, key, key_length*sizeof(int), hipMemcpyHostToDevice));
    memoryTime.stop();

    // execute kernel
    kernelTime1.start();
    if (n % threadBlockSize == 0) {
        decryptKernel<<<n/threadBlockSize, threadBlockSize>>>(deviceDataIn, deviceDataOut, deviceKey);
    } else {
        decryptKernel<<<(n/threadBlockSize) + 1, threadBlockSize>>>(deviceDataIn, deviceDataOut, deviceKey);
    }
    hipDeviceSynchronize();
    kernelTime1.stop();

    // check whether the kernel invocation was successful
    checkCudaCall(hipGetLastError());

    // copy result back
    memoryTime.start();
    checkCudaCall(hipMemcpy(data_out, deviceDataOut, n * sizeof(char), hipMemcpyDeviceToHost));
    memoryTime.stop();

    checkCudaCall(hipFree(deviceDataIn));
    checkCudaCall(hipFree(deviceDataOut));
    checkCudaCall(hipFree(deviceKey));

    cout << fixed << setprecision(6);
    cout << "Decrypt (kernel): \t\t" << kernelTime1.getElapsed() << " seconds." << endl;
    cout << "Decrypt (memory): \t\t" << memoryTime.getElapsed() << " seconds." << endl;

   return 0;
}

/* Entry point to the function! */
int main(int argc, char* argv[]) {
    // Check if there are enough arguments
    if (argc < 2) {
        cout << "Usage: " << argv[0] << " key..." << endl;
        cout << " - key: one or more values for the encryption key, separated "
                "by spaces" << endl;
        return EXIT_FAILURE;
    }

    // Parse the keys from the command line arguments
    int key_length = argc - 1;
    int *enc_key = new int[key_length];
    for (int i = 0; i < key_length; i++) {
        enc_key[i] = atoi(argv[i + 1]);
    }

    // Check if the original.data file exists and what it's size is
    int n;
    n = fileSize("10mb.txt");
    if (n == -1) {
        cout << "File not found! Exiting ... " << endl;
        exit(0);
    }


    // Read the file in memory from the disk
    char* data_in = new char[n];
    char* data_out = new char[n];
    readData("10mb.txt", data_in);

    cout << "Encrypting a file of " << n << " characters." << endl;

    EncryptSeq(n, data_in, data_out, key_length, enc_key);
    writeData(n, "sequential.data", data_out);
    EncryptCuda(n, data_in, data_out, key_length, enc_key);
    writeData(n, "cuda.data", data_out);

    readData("cuda.data", data_in);

    cout << "Decrypting a file of " << n << "characters" << endl;
    DecryptSeq(n, data_in, data_out, key_length, enc_key);
    writeData(n, "sequential_recovered.data", data_out);
    DecryptCuda(n, data_in, data_out, key_length, enc_key);
    writeData(n, "recovered.data", data_out);

    delete[] data_in;
    delete[] data_out;

    return 0;
}
