#include "hip/hip_runtime.h"
/*
 * simulate.cu
 *
 * Implementation of a wave equation simulation, parallelized on the GPU using
 * CUDA.
 *
 * You are supposed to edit this file with your implementation, and this file
 * only.
 *
 */

 #include <stdio.h>
 #include <stdlib.h>
 #include <math.h>
 #include <string.h>
 #include "timer.hh"
 #include <iostream>

#include "simulate.hh"

using namespace std;
__constant__ double c = 0.15;
__constant__ long max_i = 1000000;


/* Utility function, use to do error checking for CUDA calls
 *
 * Use this function like this:
 *     checkCudaCall(<cuda_call>);
 *
 * For example:
 *     checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));
 *
 * Special case to check the result of the last kernel invocation:
 *     kernel<<<...>>>(...);
 *     checkCudaCall(hipGetLastError());
**/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(EXIT_FAILURE);
    }
}


__global__ void wave_eq_Kernel(double *old_array, double *current_array, double *next_array) {
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i > 0 and i < max_i-1) {
        next_array[i] = 2 * current_array[i] - old_array[i] + c * (current_array[i - 1] - (2 * current_array[i] - current_array[i + 1]));
    }
    double* temp = old_array;
    old_array = current_array;
    current_array = next_array;
    next_array = temp;
}

/* Function that will simulate the wave equation, parallelized using CUDA.
 *
 * i_max: how many data points are on a single wave
 * t_max: how many iterations the simulation should run
 * num_threads: how many threads to use (excluding the main threads)
 * old_array: array of size i_max filled with data for t-1
 * current_array: array of size i_max filled with data for t
 * next_array: array of size i_max. You should fill this with t+1
 */
double *simulate(const long i_max, const long t_max, const long block_size,
                 double *old_array, double *current_array, double *next_array) {
    int threadBlockSize = 512;
    

    float* deviceA = NULL;
    checkCudaCall(hipMalloc((void **) &deviceA, i_max * sizeof(double)));
    if (deviceA == NULL) {
        cerr << "Error allocating memory for a on the device" << endl;
        return 0;
    }

    float* deviceB = NULL;
    checkCudaCall(hipMalloc((void **) &deviceB, i_max * sizeof(double)));
    if (deviceB == NULL) {
        checkCudaCall(hipFree(deviceA));
        cerr << "Error allocating memory for B on the device" << endl;
        return 0;
    }

    float* deviceC = NULL;
    checkCudaCall(hipMalloc((void **) &deviceC, i_max * sizeof(double)));
    if (deviceC == NULL) {
        checkCudaCall(hipFree(deviceA));
        checkCudaCall(hipFree(deviceB));
        cerr << "Error allocating memory for C on the device" << endl;
        return 0;
    }

    cout << max_i/threadBlockSize << endl;
    //CUDA timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    for (int t = 0; t < t_max; t++) {
        // Copy the original arrays to the GPU
        checkCudaCall(hipMemcpy(deviceA, old_array, i_max*sizeof(double), hipMemcpyHostToDevice));
        checkCudaCall(hipMemcpy(deviceB, current_array, i_max*sizeof(double), hipMemcpyHostToDevice));
        checkCudaCall(hipMemcpy(deviceC, next_array, i_max*sizeof(double), hipMemcpyHostToDevice));

        // Execute the wave_eq_kernel
        hipEventRecord(start, 0);
        
        wave_eq_Kernel<<<max_i/threadBlockSize, threadBlockSize>>>(deviceA, deviceB, deviceC);
        hipEventRecord(stop, 0);

        // Check whether the kernel invocation was successful
        checkCudaCall(hipGetLastError());
    }
    // Copy result back to host
    checkCudaCall(hipMemcpy(old_array, deviceA, i_max*sizeof(double), hipMemcpyDeviceToHost));
    checkCudaCall(hipMemcpy(current_array, deviceB, i_max*sizeof(double), hipMemcpyDeviceToHost));
    checkCudaCall(hipMemcpy(next_array, deviceC, i_max*sizeof(double), hipMemcpyDeviceToHost));

    // Cleanup GPU-side data
    checkCudaCall(hipFree(deviceA));
    checkCudaCall(hipFree(deviceB));
    checkCudaCall(hipFree(deviceC));

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "Kernel invocation took " << elapsedTime << " milliseconds" << endl;

    return current_array;
}
