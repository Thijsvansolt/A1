#include "hip/hip_runtime.h"
/*
 * simulate.cu
 *
 * Implementation of a wave equation simulation, parallelized on the GPU using
 * CUDA.
 *
 * You are supposed to edit this file with your implementation, and this file
 * only.
 *
 */

 #include <stdio.h>
 #include <stdlib.h>
 #include <math.h>
 #include <string.h>
 #include "timer.hh"
 #include <iostream>

#include "simulate.hh"

using namespace std;
__constant__ double c = 0.15;



/* Utility function, use to do error checking for CUDA calls
 *
 * Use this function like this:
 *     checkCudaCall(<cuda_call>);
 *
 * For example:
 *     checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));
 *
 * Special case to check the result of the last kernel invocation:
 *     kernel<<<...>>>(...);
 *     checkCudaCall(hipGetLastError());
**/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(EXIT_FAILURE);
    }
}


__global__ void wave_eq_Kernel(double *old_array, double *current_array, double *next_array) {
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i > 0 and i < max_domain - 1) {
        next_array[i] = 2 * current_array[i] - old_array[i] + c * (current_array[i - 1] - (2 * current_array[i] - current_array[i + 1]));
    }
    double* temp = old_array;
    old_array = current_array;
    current_array = next_array;
    next_array = temp;
}

/* Function that will simulate the wave equation, parallelized using CUDA.
 *
 * i_max: how many data points are on a single wave
 * t_max: how many iterations the simulation should run
 * num_threads: how many threads to use (excluding the main threads)
 * old_array: array of size i_max filled with data for t-1
 * current_array: array of size i_max filled with data for t
 * next_array: array of size i_max. You should fill this with t+1
 */
double *simulate(const long i_max, const long t_max, const long block_size,
                 double *old_array, double *current_array, double *next_array) {
    int threadBlockSize = 512;
    __constant__ long max_domain = i_max;

    double* old = NULL;
    checkCudaCall(hipMalloc((void **) &old, i_max * sizeof(double)));
    if (old == NULL) {
        cerr << "Error allocating memory for a on the device" << endl;
        return 0;
    }

    double* current = NULL;
    checkCudaCall(hipMalloc((void **) &current, i_max * sizeof(double)));
    if (current == NULL) {
        checkCudaCall(hipFree(old));
        cerr << "Error allocating memory for B on the device" << endl;
        return 0;
    }

    double* next = NULL;
    checkCudaCall(hipMalloc((void **) &next, i_max * sizeof(double)));
    if (next == NULL) {
        checkCudaCall(hipFree(old));
        checkCudaCall(hipFree(current));
        cerr << "Error allocating memory for C on the device" << endl;
        return 0;
    }

    // cout << max_i/threadBlockSize << endl;
    //CUDA timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    for (int t = 0; t < t_max; t++) {
        // Copy the original arrays to the GPU
        checkCudaCall(hipMemcpy(old, old_array, i_max*sizeof(double), hipMemcpyHostToDevice));
        checkCudaCall(hipMemcpy(current, current_array, i_max*sizeof(double), hipMemcpyHostToDevice));
        checkCudaCall(hipMemcpy(next, next_array, i_max*sizeof(double), hipMemcpyHostToDevice));

        // Execute the wave_eq_kernel
        hipEventRecord(start, 0);
        wave_eq_Kernel<<<i_max/threadBlockSize, threadBlockSize>>>(old, current, next);
        hipEventRecord(stop, 0);

        // Check whether the kernel invocation was successful
        checkCudaCall(hipGetLastError());
    }
    // Copy result back to host
    checkCudaCall(hipMemcpy(old_array, old, i_max*sizeof(double), hipMemcpyDeviceToHost));
    checkCudaCall(hipMemcpy(current_array, current, i_max*sizeof(double), hipMemcpyDeviceToHost));
    checkCudaCall(hipMemcpy(next_array, next, i_max*sizeof(double), hipMemcpyDeviceToHost));

    // Cleanup GPU-side data
    checkCudaCall(hipFree(old));
    checkCudaCall(hipFree(current));
    checkCudaCall(hipFree(next));

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "Kernel invocation took " << elapsedTime << " milliseconds" << endl;

    return current_array;
}
